#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>



__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    printf("%f",stride);
    for(int i = index; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void initWith(float num, float *a, int N)
{
    for(int i = 0; i < N; ++i)
    {
        a[i] = num;
    }
}


void checkElementsAre(float target, float *vector, int N)
{
    for(int i = 0; i < N; i++)
    {
        if(vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main(){
    std::ofstream save;

    std::chrono::system_clock::time_point start;
    std::chrono::system_clock::time_point stop;
    std::chrono::duration<double> elapsed_time;

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    std::cout<<"Device ID: " << deviceId << "Number of SMs: " <<numberOfSMs<<std::endl;

    const int N = 2<<24;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;


    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWith(3, a, N);
    initWith(4, b, N);
    initWith(0, c, N);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    hipError_t addVectorsErr;
    hipError_t asyncErr;

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    checkElementsAre(7, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
